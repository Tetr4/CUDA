#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include "kernel.h"
#include <limits.h>
#include <math.h>


/**
 * The inner computational kernel (will be executed on the GPU).
 *
 * @param       lbDev   Left image (global memory)
 * @param       rbDev   Right image (global memory)
 * @param	zeilen	number of rows of the image
 * @param	spalten	number of columns of the image
 * @param       b       width of the correlation window
 * @param       h       height of the correlation window
 * @param       tauMax  Maximal dispersion
 * @param       useS    Use a threshold (Schwellwert)
 * @param       useF    Use weighted window (please ignore)
 * @param       s       threshold to apply (when useS==true)
 * @param       profileDev Dispersion profile in global memory (to be filled by the algorithm)
 * @param       validDev   Validation matrix in global memory (to be filled by the algorithm)
 */
__global__ void StereoKernel(char *lbDev,char *rbDev, int zeilen, int spalten,int tauMax, int *profileDev,
	      unsigned char *validDev, int b, int h, bool useS, bool useF, int s)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if(i >= spalten | j >= zeilen)
	{
		// out of bounds
		return;
	}

	int xu = -b / 2;
	int xo = b / 2;
	int yu = -h / 2;
	int yo = h / 2;
	
	int index = i + j * spalten;

	if ((i + xu - tauMax < 0) | (i + xo + tauMax >= spalten) | (j + yu < 0) | (j + yo >= zeilen))
	{
		// invalid
		profileDev[index] = 0;
		validDev[index] = 0;
	}
	else 
	{
		int optWert = INT_MAX;
		int optIndex = 0;
		int val = 0;
		int wert;
		int windowIndex;
		for (int tau = -tauMax; tau <= tauMax; tau++)
		{
			wert = 0;
			for (int k = xu; k <= xo; k++)
			{
				for (int l = yu; l <= yo; l++)
				{
					windowIndex = i+k + (j+l)*spalten;
					wert += abs( (unsigned char) lbDev[windowIndex] - (unsigned char) rbDev[windowIndex +tau] );
				}
			}
			if (wert < optWert)
			{
				optWert = wert;
				optIndex = tau;
				val = 1;
			}
			else if (wert == optWert)
			{
				val = 0;
			}
		}
		if ((useS) && (optWert > s))
		{
			// not in treshold
			val = 0;
		}
		profileDev[index] = optIndex;
		validDev[index] = val;
	}
}

/**
 * Perform the stereodiff algorithm in CUDA.
 *
 * @param       lb      Left image
 * @param       rb      Right image
 * @param	zeilen	number of rows of the image
 * @param	spalten	number of columns of the image
 * @param       b       width of the correlation window
 * @param       h       height of the correlation window
 * @param       tauMax  Maximal dispersion
 * @param       useS    Use a threshold (Schwellwert)
 * @param       useF    Use weighted window (please ignore)
 * @param       s       threshold to apply (when useS==true)
 * @param       profile Dispersion profile (to be filled by the algorithm)
 * @param       valid  Validation matrix (to be filled by the algorithm)
 */
void doCUDACalc(signed char *lb, signed char *rb, int zeilen, int spalten,int tauMax, int *profile,
 unsigned char *valid, int b, int h, bool useS, bool useF, int s)
{


	int num_devices, device;
	hipGetDeviceCount(&num_devices);
	if (num_devices > 1) {
      		int max_multiprocessors = 0, max_device = 0;
      		for (device = 0; device < num_devices; device++) {
              		hipDeviceProp_t properties;
              		hipGetDeviceProperties(&properties, device);
              		if (max_multiprocessors < properties.multiProcessorCount) {
                      		max_multiprocessors = properties.multiProcessorCount;
                      		max_device = device;
              		}
      		}
      		hipSetDevice(max_device);
	}

	char *lbDev; // left image in CUDA memory
	char *rbDev; // right image in CUDA memory
	int *profileDev; // profile matrix. to be filled by the algorithm
	unsigned char *validDev; // valid matrix. to be filled by the algorithm

	// allocate global memory for the GPU
	checkCudaErrors(hipMalloc((void**)&lbDev, zeilen*spalten*sizeof(char)));
	checkCudaErrors(hipMalloc((void**)&rbDev, zeilen*spalten*sizeof(char)));
	checkCudaErrors(hipMalloc((void**)&profileDev, zeilen*spalten*sizeof(int)));
	checkCudaErrors(hipMalloc((void**)&validDev, zeilen*spalten*sizeof(unsigned char)));

	printf("Copying data to CUDA memory\n");
	checkCudaErrors(hipMemcpy(lbDev, lb, spalten*zeilen, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(rbDev, rb, spalten*zeilen, hipMemcpyHostToDevice));


	// call the kernel
	dim3 dimBlock (16, 16);
	dim3 dimGrid( (spalten + dimBlock.x - 1) / dimBlock.x, (zeilen + dimBlock.y -1) / dimBlock.y );
	StereoKernel<<< dimGrid , dimBlock >>>(lbDev, rbDev, zeilen, spalten, tauMax, profileDev, validDev, b, h, useS, useF, s) ;


	printf("Copying results back from CUDA memory\n");
	checkCudaErrors(hipMemcpy(valid, validDev, spalten*zeilen, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(profile, profileDev, spalten*zeilen*sizeof(int), hipMemcpyDeviceToHost));

	// free memory
	checkCudaErrors(hipFree(lbDev));
	checkCudaErrors(hipFree(rbDev));
	checkCudaErrors(hipFree(profileDev));
	checkCudaErrors(hipFree(validDev));

        hipDeviceReset();

}


